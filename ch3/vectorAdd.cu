#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

int main() {
    hipError_t err = hipSuccess;
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elementes]\n", numElements);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (int i  =0; i < numElements; ++i) {
        h_A[i] = rand() / (float) RAND_MAX;
        h_B[i] = rand() / (float) RAND_MAX;
    }

    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
    threadsPerBlock);
    vectorAdd <<< blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements);
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
        }
      }
    

      printf("Test PASSED\n");

      // Free device global memory
      err = hipFree(d_A);
    
      if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
      }
    
      err = hipFree(d_B);
    
      if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
      }
    
      err = hipFree(d_C);
    
      if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
      }
    
      // Free host memory
      free(h_A);
      free(h_B);
      free(h_C);
    
      printf("Done\n");
      return 0;
}